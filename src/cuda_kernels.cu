#include "hip/hip_runtime.h"
#include "cuda_kernels.h"
#include "vec3_struct.hpp"
#include "triangle_struct.hpp"
#include "viewpoint_struct.hpp"

#include <vector>
#include <limits>
#include <cmath>

// one origin, mapped to many end points
__global__ void ray_int_plane(
    bool *result, // flattened 3d
    vec3 *int_points, // flattened 3d
    const vec3 origin,  // vp (vec3)
    const vec3 *ends,    // n_vp (1dim)
    const Triangle *tri,// n_tri (1dim)
    size_t n_tri
    ) {

    // epsilon for floating point comparison
    float eps = 1e-6f;

    // get indices
    size_t vp_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_tri
    size_t tri_idx = blockIdx.y * blockDim.y + threadIdx.y; // n_tri
    size_t tri_pt_idx = blockIdx.z * blockDim.z + threadIdx.z; // 3
    size_t ray_idx = vp_idx * 3 + tri_pt_idx; // n_ray
    size_t res_idx = tri_pt_idx * n_tri * n_tri + tri_idx * n_tri + vp_idx;

    if (vp_idx > n_tri - 1 || tri_idx > n_tri - 1 || tri_pt_idx > 2) { return; }


    // instantiate ray
    vec3 end = ends[ray_idx];
    vec3 vec = end - origin;

    // look for any intersections between the ray and triangle
    vec3 e1 = tri[tri_idx].b - tri[tri_idx].a;
    vec3 e2 = tri[tri_idx].c - tri[tri_idx].a;
    vec3 h = vec.cross(e2);
    float a = e1.dot(h);

    // if ray is parallel to triangle
    if (a > -eps && a < eps) {
        result[res_idx] = false;
        return;
    }

    float f = 1 / a;
    vec3 s = origin - tri[tri_idx].a;
    float u = f * s.dot(h);
    if (u < 0.0f || u > 1.0f) {
        result[res_idx] = false;
        return;
    }
    vec3 q = s.cross(e1);
    float v = f * vec.dot(q);
    if (v < 0.0f || u + v > 1.0f) {
        result[res_idx] = false;
        return;
    }

    // find intersection point
    float t = f * e2.dot(q);
    vec3 intPoint = origin + vec * t;
    int_points[res_idx] = intPoint;

    // check if intersection point is between origin and end
    vec3 vec_dir = vec/vec.norm();
    if ((intPoint-origin).dot(vec_dir) < vec.norm() - eps && (intPoint-origin).dot(vec_dir) > 0) {
        result[res_idx] = true;
        return;
    }

    result[res_idx] = false;
    return;
}

// dims: viewpoints (x dim) x faces (y dim) x 3 (tri dim)
// many origins, each mapped to an end point
__global__ void ray_int_plane_many(
    bool *result, // flattened 3d
    vec3 *int_points, // flattened 3d
    const vec3 *starts,  // n_vp (vec3)
    const vec3 *ends,    // n_vp (1dim)
    size_t n_vp,
    const Triangle *tri,// n_tri (1dim)
    size_t n_tri
    ) {

    // epsilon for floating point comparison
    float eps = 1e-6f;

    // get indices
    size_t vp_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_tri
    size_t tri_idx = blockIdx.y * blockDim.y + threadIdx.y; // n_tri
    size_t tri_pt_idx = blockIdx.z * blockDim.z + threadIdx.z; // 3
    size_t ray_idx = vp_idx * 3 + tri_pt_idx; // n_ray
    size_t res_idx = tri_pt_idx * n_tri * n_vp + tri_idx * n_vp + vp_idx;

    if (vp_idx > n_vp - 1 || tri_idx > n_tri - 1 || tri_pt_idx > 2) { return; }


    // instantiate ray
    vec3 origin = starts[ray_idx];
    vec3 end = ends[ray_idx];
    vec3 vec = end - origin;

    // look for any intersections between the ray and triangle
    vec3 e1 = tri[tri_idx].b - tri[tri_idx].a;
    vec3 e2 = tri[tri_idx].c - tri[tri_idx].a;
    vec3 h = vec.cross(e2);
    float a = e1.dot(h);

    // if ray is parallel to triangle
    if (a > -eps && a < eps) {
        result[res_idx] = false;
        return;
    }

    float f = 1 / a;
    vec3 s = origin - tri[tri_idx].a;
    float u = f * s.dot(h);
    if (u < 0.0f || u > 1.0f) {
        result[res_idx] = false;
        return;
    }
    vec3 q = s.cross(e1);
    float v = f * vec.dot(q);
    if (v < 0.0f || u + v > 1.0f) {
        result[res_idx] = false;
        return;
    }

    // find intersection point
    float t = f * e2.dot(q);
    vec3 intPoint = origin + vec * t;
    int_points[res_idx] = intPoint;

    // check if intersection point is between origin and end
    vec3 vec_dir = vec/vec.norm();
    if ((intPoint-origin).dot(vec_dir) < vec.norm() - eps && (intPoint-origin).dot(vec_dir) > 0) {
        result[res_idx] = true;
        return;
    }

    result[res_idx] = false;
    return;
}

__global__ void collision_or(bool* vp_collision, const bool* ray_tri_collision, size_t n_vp, size_t n_tri) {
    // for each viewpoint-triangle correspondance, check if rays to each vertex collide with any other triangle. if so, write in true
    // get viewpoint index
    size_t vp_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_vp

    if (vp_idx > n_vp - 1) { return; } // n_tri = n_vp

    vp_collision[vp_idx] = false;
    for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
        for (size_t tri_pt_idx = 0; tri_pt_idx < 3; tri_pt_idx++) {
            size_t res_idx = tri_pt_idx * n_tri * n_vp + tri_idx * n_vp + vp_idx;
            if (ray_tri_collision[res_idx]) {
                vp_collision[vp_idx] = true;
            }
        }
    }
    return;
}

__global__ void inc_angle(
    float *angles, // flattened 2d
    const vec3 *poses, // n_vp (vec3)
    const vec3 *centroids, // n_tri (vec3)
    const vec3 *normals, // n_tri (vec3)
    size_t n_vp,
    size_t n_tri
    ) {

    // get indices
    size_t vp_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_vp
    size_t tri_idx = blockIdx.y * blockDim.y + threadIdx.y; // n_tri
    size_t res_idx = tri_idx * n_vp + vp_idx; // n_tri * n_vp

    if (vp_idx > n_vp - 1 || tri_idx > n_tri - 1) { return; }

    // calculate angle
    vec3 vec = centroids[tri_idx] - poses[vp_idx];
    vec3 norm = normals[tri_idx];
    float angle = acos(fabsf(vec.dot(norm))/(vec.norm()*norm.norm()));
    angles[res_idx] = angle;
    return;
}

extern "C" void cuda_kernel_coverage(
    const Viewpoint& vp, 
    const std::vector<Triangle*>& faces,
    std::vector<bool>& collisions,
    vec3** int_points
    ) {

    // put viewpoints into arrays
    size_t n_tri = faces.size();
    size_t n_ray = n_tri * 3;

    vec3 *ends = new vec3[n_ray];
    Triangle *tri = new Triangle[n_tri];
    bool *result_arr = new bool[n_tri];
    bool *intersection_arr = new bool[n_ray * n_tri];
    vec3 *result_int_points = new vec3[n_ray * n_tri];

    // thread, block size
    size_t thread_x = 16;
    size_t thread_y = 16;
    size_t thread_z = 3;

    // put faces into array
    for (size_t i = 0; i < n_tri; i++) {
        tri[i] = *faces[i];
    }

    // put viewpoints into array
    for (size_t tri_pt_idx = 0; tri_pt_idx < 3; tri_pt_idx++) {
        for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
            size_t ray_idx = tri_idx * 3 + tri_pt_idx; // n_ray
            if (tri_pt_idx == 0) {
                ends[ray_idx] = tri[tri_idx].a;
            } else if (tri_pt_idx == 1) {
                ends[ray_idx] = tri[tri_idx].b;
            } else if (tri_pt_idx == 2) {
                ends[ray_idx] = tri[tri_idx].c;
            }
        }
    }

    // allocate gpu memory
    vec3 *d_ends;
    Triangle *d_tri;

    bool *d_intersections; // collisions per ray per triangle
    bool *d_result; // collisions per triangle
    vec3 *d_int_points; // intersection points

    hipMalloc(&d_ends, n_ray * sizeof(vec3));
    hipMalloc(&d_tri, n_tri * sizeof(Triangle));
    hipMalloc(&d_result, n_tri * sizeof(bool));
    hipMalloc(&d_intersections, n_ray * n_tri * sizeof(bool));
    hipMalloc(&d_int_points, n_ray * n_tri * sizeof(vec3));

    // copy data to gpu
    hipMemcpy(d_ends, ends, n_ray * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_tri, tri, n_tri * sizeof(Triangle), hipMemcpyHostToDevice);

    // set thread and block size
    dim3 threadsPerBlock(thread_x, thread_y, thread_z);

    // 2D blocks because 3d blocks can account for the 3rd dim by themselves
    dim3 numBlocks(int((n_tri + threadsPerBlock.x - 1) / threadsPerBlock.x), (n_tri + threadsPerBlock.y - 1) / threadsPerBlock.y, 1);
    ray_int_plane<<<numBlocks, threadsPerBlock>>>(d_intersections, d_int_points, vp.pose, d_ends, d_tri, n_tri);

    hipMemcpy(intersection_arr, d_intersections, n_ray * n_tri * sizeof(bool), hipMemcpyDeviceToHost);

    // same as above but without the 3rd dimension
    threadsPerBlock.x = 1024;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;
    numBlocks.x = (n_tri + threadsPerBlock.x - 1) / threadsPerBlock.x;
    collision_or<<<numBlocks, threadsPerBlock>>>(d_result, d_intersections, n_tri, n_tri);

    hipMemcpy(result_arr, d_result, n_tri * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(result_int_points, d_int_points, n_ray * n_tri * sizeof(vec3), hipMemcpyDeviceToHost);

    hipFree(d_ends);
    hipFree(d_tri);
    hipFree(d_result);
    hipFree(d_intersections);
    hipFree(d_int_points);

    for (size_t vp_idx=0; vp_idx < n_tri; vp_idx++) {
        collisions.push_back(result_arr[vp_idx]);
    }

    if (int_points != nullptr) {
        for (size_t vp_idx = 0; vp_idx < n_tri; vp_idx++) {
            for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
                for (size_t tri_pt_idx = 0; tri_pt_idx < 3; tri_pt_idx++) {
                    size_t res_idx = tri_pt_idx * n_tri * n_tri + tri_idx * n_tri + vp_idx;
                    size_t ray_idx = tri_idx * 3 + tri_pt_idx;
                    if (intersection_arr[res_idx]) {
                        int_points[vp_idx][ray_idx] = result_int_points[res_idx];
                    } else {
                        int_points[vp_idx][ray_idx] = vec3(
                            std::numeric_limits<float>::infinity(), 
                            std::numeric_limits<float>::infinity(), 
                            std::numeric_limits<float>::infinity()
                        );
                    }
                    // int_points[vp_idx][ray_idx] = result_int_points[res_idx];
                }
            }
        }
    }

    delete[] ends;
    delete[] tri;
    delete[] result_arr;
    delete[] intersection_arr;
    delete[] result_int_points;
}

extern "C" void cuda_kernel_many(
    const std::vector<Viewpoint>& viewpoints,
    const std::vector<size_t>& triangle_indices,
    const std::vector<Triangle*>& faces,
    bool* collisions,
    vec3** int_points
    ) {

    // put viewpoints into arrays
    size_t n_tri = faces.size();
    size_t n_vp = viewpoints.size();
    size_t n_ray = n_vp * 3;

    vec3 *starts = new vec3[n_ray];
    vec3 *ends = new vec3[n_ray];
    Triangle *tri = new Triangle[n_tri];
    bool *result_arr = new bool[n_vp];
    bool *intersection_arr = new bool[n_ray * n_tri];
    vec3 *result_int_points = new vec3[n_ray * n_tri];

    // thread, block size
    size_t thread_x = 16;
    size_t thread_y = 16;
    size_t thread_z = 3;

    // put faces into array
    for (size_t i = 0; i < n_tri; i++) {
        tri[i] = *faces[i];
    }

    // put viewpoints into array
    for (size_t tri_pt_idx = 0; tri_pt_idx < 3; tri_pt_idx++) {
        for (size_t vp_idx = 0; vp_idx < n_vp; vp_idx++) {
            size_t ray_idx = vp_idx * 3 + tri_pt_idx; // n_ray
            size_t tri_idx = triangle_indices[vp_idx];
            starts[ray_idx] = viewpoints[vp_idx].pose;
            if (tri_pt_idx == 0) {
                ends[ray_idx] = tri[tri_idx].a;
            } else if (tri_pt_idx == 1) {
                ends[ray_idx] = tri[tri_idx].b;
            } else if (tri_pt_idx == 2) {
                ends[ray_idx] = tri[tri_idx].c;
            }
        }
    }

    // allocate gpu memory
    vec3 *d_starts;
    vec3 *d_ends;
    Triangle *d_tri;

    bool *d_intersections; // collisions per ray per triangle
    bool *d_result; // collisions per triangle
    vec3 *d_int_points; // intersection points

    hipMalloc(&d_starts, n_ray * sizeof(vec3));
    hipMalloc(&d_ends, n_ray * sizeof(vec3));
    hipMalloc(&d_tri, n_tri * sizeof(Triangle));
    hipMalloc(&d_intersections, n_ray * n_tri * sizeof(bool));
    hipMalloc(&d_result, n_vp * sizeof(bool));
    hipMalloc(&d_int_points, n_ray * n_tri * sizeof(vec3));

    // copy data to gpu
    hipMemcpy(d_starts, starts, n_ray * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_ends, ends, n_ray * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_tri, tri, n_tri * sizeof(Triangle), hipMemcpyHostToDevice);

    // set thread and block size
    dim3 threadsPerBlock(thread_x, thread_y, thread_z);

    // 2D blocks because 3d blocks can account for the 3rd dim by themselves
    dim3 numBlocks(int((n_vp + threadsPerBlock.x - 1) / threadsPerBlock.x), (n_tri + threadsPerBlock.y - 1) / threadsPerBlock.y, 1);
    ray_int_plane_many<<<numBlocks, threadsPerBlock>>>(
        d_intersections, 
        d_int_points, 
        d_starts, 
        d_ends, 
        n_vp,
        d_tri, 
        n_tri
    );

    hipDeviceSynchronize();

    hipMemcpy(intersection_arr, d_intersections, n_ray * n_tri * sizeof(bool), hipMemcpyDeviceToHost);

    // same as above but without the 3rd dimension
    threadsPerBlock.x = 1024;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    // reusing numBlocks
    numBlocks.x = (n_tri + threadsPerBlock.x - 1) / threadsPerBlock.x;
    numBlocks.y = 1; // numBlocks.z is already 1
    collision_or<<<numBlocks, threadsPerBlock>>>(d_result, d_intersections, n_vp, n_tri);

    hipDeviceSynchronize();

    hipMemcpy(result_arr, d_result, n_vp * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(result_int_points, d_int_points, n_ray * n_tri * sizeof(vec3), hipMemcpyDeviceToHost);

    hipFree(d_starts);
    hipFree(d_ends);
    hipFree(d_tri);
    hipFree(d_result);
    hipFree(d_intersections);
    hipFree(d_int_points);

    for (size_t vp_idx=0; vp_idx < n_vp; vp_idx++) {
        collisions[vp_idx] = result_arr[vp_idx];
    }

    if (int_points != nullptr) {
        for (size_t vp_idx = 0; vp_idx < n_vp; vp_idx++) {
            for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
                for (size_t tri_pt_idx = 0; tri_pt_idx < 3; tri_pt_idx++) {
                    size_t res_idx = tri_pt_idx * n_tri * n_vp + tri_idx * n_vp + vp_idx;
                    size_t ray_idx = tri_idx * 3 + tri_pt_idx;
                    if (intersection_arr[res_idx]) {
                        int_points[vp_idx][ray_idx] = result_int_points[res_idx];
                    } else {
                        int_points[vp_idx][ray_idx] = vec3(
                            std::numeric_limits<float>::infinity(), 
                            std::numeric_limits<float>::infinity(), 
                            std::numeric_limits<float>::infinity()
                        );
                    }
                }
            }
        }
    }

    delete[] starts;
    delete[] ends;
    delete[] tri;
    delete[] result_arr;
    delete[] intersection_arr;
    delete[] result_int_points;
}

extern "C" void cuda_kernel_inc_angle(
    std::vector<Viewpoint>& viewpoints,
    std::vector<Triangle*>& faces,
    std::vector<std::vector<float>>& inc_angles // n_vp x n_tri
    ) {

    // put viewpoints into arrays
    size_t n_tri = faces.size();
    size_t n_vp = viewpoints.size();

    vec3 *poses = new vec3[n_vp];
    vec3 *centroids = new vec3[n_tri];
    vec3 *normals = new vec3[n_tri];
    float *angles = new float[n_vp * n_tri];

    // put faces into array
    for (size_t vp_idx = 0; vp_idx < n_vp; vp_idx++) {
        poses[vp_idx] = viewpoints[vp_idx].pose;
    }

    for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
        centroids[tri_idx] = faces[tri_idx]->getCentroid();
        normals[tri_idx] = faces[tri_idx]->n;
    }

    // initialize pointers for gpu memory
    vec3 *d_poses;
    vec3 *d_centroids;
    vec3 *d_normals;
    float *d_angles;

    // allocate memory on gpu
    hipMalloc(&d_poses, n_vp * sizeof(vec3));
    hipMalloc(&d_centroids, n_tri * sizeof(vec3));
    hipMalloc(&d_normals, n_tri * sizeof(vec3));
    hipMalloc(&d_angles, n_vp * n_tri * sizeof(float));

    // copy data to gpu
    hipMemcpy(d_poses, poses, n_vp * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids, n_tri * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_normals, normals, n_tri * sizeof(vec3), hipMemcpyHostToDevice);

    // set thread and block size
    size_t thread_x = 32;
    size_t thread_y = 32;
    size_t thread_z = 1;
    dim3 threadsPerBlock(thread_x, thread_y, thread_z);
    dim3 numBlocks(
        int((n_vp + threadsPerBlock.x - 1) / threadsPerBlock.x), 
        int((n_tri + threadsPerBlock.y - 1) / threadsPerBlock.y),
        1
    );

    // calculate incidence angles
    inc_angle<<<numBlocks, threadsPerBlock>>>(
        d_angles, 
        d_poses, 
        d_centroids, 
        d_normals, 
        n_vp, 
        n_tri
    );

    // copy data from gpu
    hipMemcpy(angles, d_angles, n_vp * n_tri * sizeof(float), hipMemcpyDeviceToHost);

    // free memory from gpu
    hipFree(d_poses);
    hipFree(d_centroids);
    hipFree(d_normals);
    hipFree(d_angles);

    // put data into inc_angles
    for (size_t vp_idx = 0; vp_idx < n_vp; vp_idx++) {
        std::vector<float> vp_angles;
        for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
            vp_angles.push_back(angles[tri_idx * n_vp + vp_idx]);
        }
        inc_angles.push_back(vp_angles);
    }

    delete[] poses;
    delete[] centroids;
    delete[] normals;
    delete[] angles;
}