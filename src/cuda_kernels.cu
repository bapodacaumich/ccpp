#include "hip/hip_runtime.h"
#include "cuda_kernels.h"
#include "vec3_struct.hpp"
#include "triangle_struct.hpp"
#include "viewpoint_struct.hpp"

#include <vector>

// dims: viewpoints (x dim) x faces (y dim) x 3 (tri dim)
__global__ void ray_int_plane(
    bool *result, // flattened 3d
    vec3 *int_points, // flattened 3d
    const vec3 origin,  // vp (vec3)
    const vec3 *ends,    // n_vp (1dim)
    const Triangle *tri,// n_tri (1dim)
    size_t n_tri
    ) {

    // epsilon for floating point comparison
    float eps = 1e-7f;

    // get indices
    size_t vp_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_tri
    size_t tri_idx = blockIdx.y * blockDim.y + threadIdx.y; // n_tri
    size_t tri_pt_idx = blockIdx.z * blockDim.z + threadIdx.z; // 3
    size_t ray_idx = vp_idx * 3 + tri_pt_idx; // n_ray
    size_t res_idx = tri_pt_idx * n_tri * n_tri + tri_idx * n_tri + vp_idx;

    if (vp_idx > n_tri - 1 || tri_idx > n_tri - 1 || tri_pt_idx > 2) { return; }


    // instantiate ray
    vec3 end = ends[ray_idx];
    vec3 vec = end - origin;

    // int_points[res_idx] = ends[ray_idx];

    // look for any intersections between the ray and triangle
    vec3 e1 = tri[tri_idx].b - tri[tri_idx].a;
    vec3 e2 = tri[tri_idx].c - tri[tri_idx].a;
    vec3 h = vec.cross(e2);
    float a = e1.dot(h);

    // if ray is parallel to triangle
    if (a > -eps && a < eps) {
        result[res_idx] = false;
        return;
    }

    float f = 1 / a;
    vec3 s = origin - tri[tri_idx].a;
    float u = f * s.dot(h);
    if (u < 0.0f || u > 1.0f) {
        result[res_idx] = false;
        return;
    }
    vec3 q = s.cross(e1);
    float v = f * vec.dot(q);
    if (v < 0.0f || u + v > 1.0f) {
        result[res_idx] = false;
        return;
    }

    // find intersection point
    float t = f * e2.dot(q);
    vec3 intPoint = origin + vec * t;
    int_points[res_idx] = intPoint;

    // check if intersection point is between origin and end
    vec3 vec_dir = vec/vec.norm();
    if ((intPoint-origin).dot(vec_dir) < vec.norm() - eps && (intPoint-origin).dot(vec_dir) > 0) {
        result[res_idx] = true;
        return;
    }

    result[res_idx] = false;
    return;
}

__global__ void collision_or(bool* collision, bool* intersections, size_t n_tri) {
    // get indices
    size_t vp_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_ray
    size_t tri_idx = blockIdx.y * blockDim.y + threadIdx.y; // n_tri
    // size_t tri_pt_idx = blockIdx.z * blockDim.z + threadIdx.z; // 3
    size_t v0_idx = 0 * n_tri * n_tri + tri_idx * n_tri + vp_idx;
    size_t v1_idx = 1 * n_tri * n_tri + tri_idx * n_tri + vp_idx;
    size_t v2_idx = 2 * n_tri * n_tri + tri_idx * n_tri + vp_idx;

    if (v0_idx >= n_tri * n_tri) { return; }

    collision[v0_idx] = intersections[v0_idx] || intersections[v1_idx] || intersections[v2_idx];

    return;
}


extern "C" void cuda_kernel_intersect_triangles(
    const Viewpoint& vp, 
    const std::vector<Triangle*>& faces,
    bool*** collisions,
    vec3*** int_points
    ) {

    // put viewpoints into arrays
    size_t n_tri = faces.size();
    size_t n_ray = n_tri * 3;

    vec3 *ends = new vec3[n_ray];
    Triangle *tri = new Triangle[n_tri];
    std::cout << "n_tri = " << n_tri << std::endl;
    bool *result_arr = new bool[n_ray * n_tri];
    vec3 *result_int_points = new vec3[n_ray * n_tri];

    // thread, block size
    size_t thread_x = 16;
    size_t thread_y = 16;
    size_t thread_z = 3;

    // put faces into array
    for (size_t i = 0; i < n_tri; i++) {
        tri[i] = *faces[i];
    }

    // put viewpoints into array
    for (size_t tri_pt_idx = 0; tri_pt_idx < 3; tri_pt_idx++) {
        for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
            size_t ray_idx = tri_idx * 3 + tri_pt_idx; // n_ray
            if (tri_pt_idx == 0) {
                ends[ray_idx] = tri[tri_idx].a;
            } else if (tri_pt_idx == 1) {
                ends[ray_idx] = tri[tri_idx].b;
            } else if (tri_pt_idx == 2) {
                ends[ray_idx] = tri[tri_idx].c;
            }
        }
    }


    // allocate gpu memory
    vec3 *d_ends;
    Triangle *d_tri;

    bool *d_intersections; // collisions per ray per triangle
    // bool *d_result; // collisions per triangle
    vec3 *d_int_points; // intersection points

    hipMalloc(&d_ends, n_ray * sizeof(vec3));
    hipMalloc(&d_tri, n_tri * sizeof(Triangle));
    hipMalloc(&d_intersections, n_ray * n_tri * sizeof(bool));
    // hipMalloc(&d_result, n_tri * n_tri * sizeof(bool));
    hipMalloc(&d_int_points, n_ray * n_tri * sizeof(vec3));

    // copy data to gpu
    hipMemcpy(d_ends, ends, n_ray * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_tri, tri, n_tri * sizeof(Triangle), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(thread_x, thread_y, thread_z);
    // std::cout << "numBlocks = " << numBlocks.x << " " << numBlocks.y << " " << numBlocks.z << std::endl;
    std::cout << "threadsPerBlock = " << threadsPerBlock.x << " " << threadsPerBlock.y << " " << threadsPerBlock.z << std::endl;
    // 2D blocks because 3d blocks can account for the 3rd dim by themselves
    dim3 numBlocks(int((n_tri + threadsPerBlock.x - 1) / threadsPerBlock.x), (n_tri + threadsPerBlock.y - 1) / threadsPerBlock.y, 1);
    ray_int_plane<<<numBlocks, threadsPerBlock>>>(d_intersections, d_int_points, vp.pose, d_ends, d_tri, n_tri);

    // // same as above but without the 3rd dimension
    // threadsPerBlock.x = 32;
    // threadsPerBlock.y = 32;
    // threadsPerBlock.z = 1;
    // collision_or<<<numBlocks, threadsPerBlock>>>(d_result, d_intersections, n_tri);

    hipMemcpy(result_arr, d_intersections, n_ray * n_tri * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(result_int_points, d_int_points, n_ray * n_tri * sizeof(vec3), hipMemcpyDeviceToHost);
    // hipMemcpy(saved_indices, d_saved_indices, thread_x * thread_y * sizeof(size_t), hipMemcpyDeviceToHost);
    // hipMemcpy(thread_indices, d_thread_indices, thread_x * thread_y * sizeof(vec3), hipMemcpyDeviceToHost);
    // hipMemcpy(result_arr, d_result, thread_x * thread_y * thread_z * sizeof(bool), hipMemcpyDeviceToHost);
    // hipMemcpy(result_int_points, d_int_points, thread_x * thread_y * thread_z * sizeof(vec3), hipMemcpyDeviceToHost);

    hipFree(d_tri);
    hipFree(d_ends);
    hipFree(d_intersections);
    hipFree(d_int_points);
    // hipFree(d_result);
    // hipFree(d_saved_indices);
    // hipFree(d_thread_indices);
    std::cout << "Ray start=" << vp.pose.toString() << std::endl;

    for (size_t tri_pt_idx = 0; tri_pt_idx < 3; tri_pt_idx++) {
        for (size_t vp_idx = 0; vp_idx < n_tri; vp_idx++) {
            for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
                // size_t saved_idx = k * thread_x * thread_y + j * thread_x + i;
                size_t ray_idx = vp_idx * 3 + tri_pt_idx;
                size_t res_idx = tri_pt_idx * n_tri * n_tri + tri_idx * n_tri + vp_idx;
                std::cout << "Ray end=" << ends[ray_idx].toString() << " | Tri z=" << tri[tri_idx].a.z;
                std::cout << " >>>> Collision=" << result_arr[res_idx];
                std::cout << " | IntPoint=" << result_int_points[res_idx].toString() << std::endl;

                //     std::cout << " oob idx=";
                //     std::cout << result_int_points[res_idx].x << std::endl;
                // }
                // std::cout << res_idx << std::endl;
                // std::cout << "kernel ran?=" << result_arr[res_idx] << " | cuda idxs =";
                // std::cout << result_int_points[res_idx].toString() << " | ";
                // size_t res_idx_cuda = result_int_points[res_idx].x * n_vp * n_tri + result_int_points[res_idx].y * n_vp + result_int_points[res_idx].z;
                // std::cout << "cuda idx =" << res_idx_cuda << std::endl;
                // collisions[i][j][k] = result_arr[res_idx];
                // int_points[i][j][k] = result_int_points[res_idx];
            }
            // collisions[i][j] = result_arr[j * n_vp + i];
        }
    }

}