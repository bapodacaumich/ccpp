#include "hip/hip_runtime.h"
#include "cuda_kernels.h"
#include "vec3_struct.hpp"
#include "triangle_struct.hpp"
#include "viewpoint_struct.hpp"

#include <vector>
#include <limits>
#include <cmath>

__device__ void pinhole_camera(
    bool& out_of_frame,  //n_points
    vec3 pose,
    vec3 viewdir,
    vec3 point,
    float hfov=M_PI/4.0f, // rad
    float vfov=M_PI/4.0f // rad
    ) {
    // calculate the angle between the view direction and the vector from the viewpoint to the intersection point
    // size_t point_idx = blockIdx.y * blockDim.y + threadIdx.y; // n_points

    // if (point_idx > n_points - 1) { return; }

    // set up variables
    // vec3 vec = point[point_idx] - pose;
    vec3 vec = point - pose;
    // vec3 viewdir = viewdir;
    float norm_dot = vec.dot(viewdir);

    // check if point is behind camera
    if (norm_dot <= 0) {
        out_of_frame = true;
        // out_of_frame[point_idx] = false;
        return;
    }

    // project point onto view plane
    float d = 1.0f; // distance from viewpoint to view plane
    float w = 2 * d * tanf(hfov/2);
    float h = 2 * d * tanf(vfov/2);
    vec3 point_proj = (vec/norm_dot - viewdir) * d;
    vec3 v_hat = vec3(
        viewdir.z * cosf( atan2f(viewdir.y, viewdir.x) ),
        viewdir.y * sinf( atan2f(viewdir.y, viewdir.x) ),
        sqrtf(viewdir.x * viewdir.x + viewdir.y * viewdir.y) // 0 to 1
    );
    vec3 u_hat = v_hat.cross(viewdir);

    // check if point is within field of view
    if (abs(point_proj.dot(u_hat)) < w/2 && abs(point_proj.dot(v_hat)) < h/2) {
        out_of_frame = false;
        // visible[point_idx] = true;
        return;
    }

    out_of_frame = true;
    // visible[point_idx] = false;
    return;
}

// __global__ void pinhole_camera(
//     bool *visible,  // n_vp x n_points
//     vec3 *pose,
//     vec3 *viewdirs,
//     size_t n_vp,
//     vec3 *point,
//     size_t n_points,
//     float hfov=M_PI/4.0f, // rad
//     float vfov=M_PI/4.0f // rad
//     ) {
//     // calculate the angle between the view direction and the vector from the viewpoint to the intersection point
//     size_t vp_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_vp
//     size_t point_idx = blockIdx.y * blockDim.y + threadIdx.y; // n_points
//     size_t res_idx = point_idx * n_vp + vp_idx; // n_points * n_vp

//     if (vp_idx > n_vp - 1 || point_idx > n_points - 1) { return; }

//     // set up variables
//     vec3 vec = point[point_idx] - pose[vp_idx];
//     vec3 viewdir = viewdirs[vp_idx];
//     float norm_dot = vec.dot(viewdir);

//     // check if point is behind camera
//     if (norm_dot <= 0) {
//         visible[res_idx] = false;
//         return;
//     }

//     // project point onto view plane
//     float d = 1.0f; // distance from viewpoint to view plane
//     float w = 2 * d * tanf(hfov/2);
//     float h = 2 * d * tanf(vfov/2);
//     vec3 point_proj = (vec/norm_dot - viewdir) * d;
//     vec3 v_hat = vec3(
//         viewdir.z * cosf( atan2f(viewdir.y, viewdir.x) ),
//         viewdir.y * sinf( atan2f(viewdir.y, viewdir.x) ),
//         sqrtf(viewdir.x * viewdir.x + viewdir.y * viewdir.y) // 0 to 1
//     );
//     vec3 u_hat = v_hat.cross(viewdir);

//     // check if point is within field of view
//     if (abs(point_proj.dot(u_hat)) < w/2 && abs(point_proj.dot(v_hat)) < h/2) {
//         visible[res_idx] = true;
//         return;
//     }

//     visible[res_idx] = false;
//     return;
// }

// one viewpoint, mapped to many end points
__global__ void ray_int_tri(
    bool *result, // flattened 3d
    vec3 *int_points, // flattened 3d
    const vec3 origin,  // vp (vec3)
    const vec3 viewdir, // vp (vec3)
    const vec3 *ends,    // n_vp (1dim)
    const Triangle *tri, // n_tri (1dim)
    size_t n_tri
    ) {
    // true for not visiblw

    // epsilon for floating point comparison
    float eps = 1e-6f;

    // get indices
    size_t vp_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_tri
    size_t tri_idx = blockIdx.y * blockDim.y + threadIdx.y; // n_tri
    size_t tri_pt_idx = blockIdx.z * blockDim.z + threadIdx.z; // 3
    size_t ray_idx = vp_idx * 3 + tri_pt_idx; // n_ray
    size_t res_idx = tri_pt_idx * n_tri * n_tri + tri_idx * n_tri + vp_idx;

    if (vp_idx > n_tri - 1 || tri_idx > n_tri - 1 || tri_pt_idx > 2) { return; }


    // instantiate ray
    vec3 end = ends[ray_idx];
    vec3 vec = end - origin;

    // check if triangle is facing away from camera
    float norm_dot = tri[tri_idx].n.dot(origin - end);
    // if (norm_dot > 0) {
    //     result[res_idx] = true;
    //     return;
    // }

    // look for any intersections between the ray and triangle
    vec3 e1 = tri[tri_idx].b - tri[tri_idx].a;
    vec3 e2 = tri[tri_idx].c - tri[tri_idx].a;
    vec3 h = vec.cross(e2);
    float a = e1.dot(h);

    // if ray is parallel to triangle
    if (a > -eps && a < eps) {
        pinhole_camera(result[res_idx], origin, viewdir, end);
        // result[res_idx] = false;
        return;
    }

    float f = 1 / a;
    vec3 s = origin - tri[tri_idx].a;
    float u = f * s.dot(h);
    if (u < 0.0f || u > 1.0f) {
        pinhole_camera(result[res_idx], origin, viewdir, end);
        // result[res_idx] = false;
        return;
    }
    vec3 q = s.cross(e1);
    float v = f * vec.dot(q);
    if (v < 0.0f || u + v > 1.0f) {
        pinhole_camera(result[res_idx], origin, viewdir, end);
        // result[res_idx] = false;
        return;
    }

    // find intersection point
    float t = f * e2.dot(q);
    if (t < eps) {
        pinhole_camera(result[res_idx], origin, viewdir, end);
        // result[res_idx] = false;
        return;
    }
    vec3 intPoint = origin + vec * t;
    int_points[res_idx] = intPoint;

    // check if intersection point is between origin and end
    vec3 vec_dir = vec/vec.norm();
    if ((intPoint-origin).dot(vec_dir) < vec.norm() - eps && 
        (intPoint-origin).dot(vec_dir) > 0
        ) {
        result[res_idx] = true;
        return;
    }

    pinhole_camera(result[res_idx], origin, viewdir, end);
    // result[res_idx] = false;
    return;
}

// dims: viewpoints (x dim) x faces (y dim)
// many origins, each mapped to an end point
__global__ void ray_int_tri_many_2d(
    bool *result, // flattened 2d
    vec3 *int_points, // flattened 2d
    const vec3 *starts,  // n_vp (vec3)
    // const vec3 *viewdirs, // n_vp (vec3)
    const vec3 *ends,    // n_vp (vec3)
    size_t n_vp,
    const Triangle *tri,// n_tri (1dim)
    size_t n_tri
    ) {

    // epsilon for floating point comparison
    float eps = 1e-6f;

    // get indices
    size_t vp_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_tri
    size_t tri_idx = blockIdx.y * blockDim.y + threadIdx.y; // n_tri
    size_t res_idx = tri_idx * n_vp + vp_idx; // n_tri * n_vp

    if (vp_idx > n_vp - 1 || tri_idx > n_tri - 1) { return; }

    // instantiate ray
    vec3 origin = starts[vp_idx];
    vec3 end = ends[vp_idx];
    vec3 vec = end - origin;

    // look for any intersections between the ray and triangle
    vec3 e1 = tri[tri_idx].b - tri[tri_idx].a;
    vec3 e2 = tri[tri_idx].c - tri[tri_idx].a;
    vec3 h = vec.cross(e2);
    float a = e1.dot(h);

    // if ray is parallel to triangle
    if (a > -eps && a < eps) {
        result[res_idx] = false;
        return;
    }

    float f = 1 / a;
    vec3 s = origin - tri[tri_idx].a;
    float u = f * s.dot(h);
    if (u < 0.0f || u > 1.0f) {
        result[res_idx] = false;
        return;
    }
    vec3 q = s.cross(e1);
    float v = f * vec.dot(q);
    if (v < 0.0f || u + v > 1.0f) {
        result[res_idx] = false;
        return;
    }

    // find intersection point
    float t = f * e2.dot(q);
    if (t < eps) {
        result[res_idx] = false;
        return;
    }

    vec3 intPoint = origin + vec * t;
    int_points[res_idx] = intPoint;

    // check if intersection point is between origin and end
    vec3 vec_dir = vec/vec.norm();
    if ((intPoint-origin).dot(vec_dir) < vec.norm() - eps && (intPoint-origin).dot(vec_dir) > 0) {
        result[res_idx] = true;
        return;
    }

    result[res_idx] = false;
    return;
}

// dims: viewpoints (x dim) x faces (y dim) x 3 (tri dim)
// many origins, each mapped to an end point
__global__ void ray_int_tri_many(
    bool *result, // flattened 3d
    vec3 *int_points, // flattened 3d
    const vec3 *starts,  // n_ray (vec3)
    const vec3 *viewdirs, // n_vp (vec3)
    const vec3 *ends,    // n_ray (1dim)
    size_t n_vp,
    const Triangle *tri,// n_tri (1dim)
    size_t n_tri
    ) {

    // epsilon for floating point comparison
    float eps = 1e-6f;

    // get indices
    size_t vp_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_tri
    size_t tri_idx = blockIdx.y * blockDim.y + threadIdx.y; // n_tri
    size_t tri_pt_idx = blockIdx.z * blockDim.z + threadIdx.z; // 3
    size_t ray_idx = vp_idx * 3 + tri_pt_idx; // n_ray
    size_t res_idx = tri_pt_idx * n_tri * n_vp + tri_idx * n_vp + vp_idx;

    if (vp_idx > n_vp - 1 || tri_idx > n_tri - 1 || tri_pt_idx > 2) { return; }


    // instantiate ray
    vec3 origin = starts[ray_idx];
    vec3 viewdir = viewdirs[ray_idx];
    vec3 end = ends[ray_idx];
    vec3 vec = end - origin;

    // look for any intersections between the ray and triangle
    vec3 e1 = tri[tri_idx].b - tri[tri_idx].a;
    vec3 e2 = tri[tri_idx].c - tri[tri_idx].a;
    vec3 h = vec.cross(e2);
    float a = e1.dot(h);

    // if ray is parallel to triangle
    if (a > -eps && a < eps) {
        pinhole_camera(result[res_idx], origin, viewdir, end);
        // result[res_idx] = false;
        return;
    }

    float f = 1 / a;
    vec3 s = origin - tri[tri_idx].a;
    float u = f * s.dot(h);
    if (u < 0.0f || u > 1.0f) {
        pinhole_camera(result[res_idx], origin, viewdir, end);
        // result[res_idx] = false;
        return;
    }
    vec3 q = s.cross(e1);
    float v = f * vec.dot(q);
    if (v < 0.0f || u + v > 1.0f) {
        pinhole_camera(result[res_idx], origin, viewdir, end);
        // result[res_idx] = false;
        return;
    }

    // find intersection point
    float t = f * e2.dot(q);

    if (t < eps) {
        pinhole_camera(result[res_idx], origin, viewdir, end);
        // result[res_idx] = false;
        return;
    }

    vec3 intPoint = origin + vec * t;
    int_points[res_idx] = intPoint;

    // check if intersection point is between origin and end
    vec3 vec_dir = vec/vec.norm();
    if ((intPoint-origin).dot(vec_dir) < vec.norm() - eps && (intPoint-origin).dot(vec_dir) > 0) {
        result[res_idx] = true;
        return;
    }

    pinhole_camera(result[res_idx], origin, viewdir, end);
    // result[res_idx] = false;
    return;
}

__global__ void ray_int_plane(
    bool *result, // n_rays
    vec3 *int_points, // n_rays
    vec3 *ray_starts,  // n_rays (vec3)
    vec3 *ray_ends,   // n_rays (vec3)
    vec3 plane_point,
    vec3 plane_normal,
    size_t n_rays
    ) {
    // get ray index 
    size_t ray_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_rays

    // check if out of bounds on rays
    if (ray_idx > n_rays - 1) { return; }

    // get ray vecs to test
    vec3 origin_to_point = plane_point - ray_starts[ray_idx];
    vec3 end_to_point = plane_point - ray_ends[ray_idx];

    // get projections onto plane normal
    float origin_proj = origin_to_point.dot(plane_normal);
    float end_proj = end_to_point.dot(plane_normal);
    float abs_origin_proj = fabsf(origin_proj);
    float abs_end_proj = fabsf(end_proj);

    // test if ray intersects plane (end and origin are on opposite sides of plane)
    if (abs_origin_proj < 1e-6f || abs_end_proj < 1e-6f) {
        result[ray_idx] = false;
        int_points[ray_idx] = vec3(
            INFINITY,
            INFINITY,
            INFINITY
        );
        return;
    }

    // must be opposite signs if they are on opposite sides of the plane
    if (origin_proj * end_proj > 0) {
        result[ray_idx] = false;
        int_points[ray_idx] = vec3(
            INFINITY,
            INFINITY,
            INFINITY
        );
        return;
    }
    float fac = abs_origin_proj / (abs_origin_proj + abs_end_proj);
    int_points[ray_idx] = ray_starts[ray_idx] * (1 - fac) + ray_ends[ray_idx] * fac;
    result[ray_idx] = true;
}

__global__ void collision_odd(bool* vp_collision, const bool* ray_tri_collision, size_t n_vp, size_t n_tri) {
    // for each viewpoint-triangle correspondance, check if rays to each vertex collide with any other triangle. if so, write in true
    // get viewpoint index
    size_t vp_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_vp

    if (vp_idx > n_vp - 1) { return; } // n_tri = n_vp

    size_t count = 0;
    vp_collision[vp_idx] = false;
    for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
        size_t res_idx = tri_idx * n_vp + vp_idx;
        if (ray_tri_collision[res_idx]) {
            count++;
        }
    }
    if ((count % 2) == 1) {
        vp_collision[vp_idx] = true;
    } else {
        vp_collision[vp_idx] = false;
    }
    return;
}

__global__ void collision_or(bool* vp_collision, const bool* ray_tri_collision, size_t n_vp, size_t n_tri) {
    // for each viewpoint-triangle correspondance, check if rays to each vertex collide with any other triangle. if so, write in true
    // get viewpoint index
    size_t vp_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_vp

    if (vp_idx > n_vp - 1) { return; } // n_tri = n_vp

    vp_collision[vp_idx] = false;
    for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
        for (size_t tri_pt_idx = 0; tri_pt_idx < 3; tri_pt_idx++) {
            size_t res_idx = tri_pt_idx * n_tri * n_vp + tri_idx * n_vp + vp_idx;
            if (ray_tri_collision[res_idx]) {
                vp_collision[vp_idx] = true;
            }
        }
    }
    return;
}

__global__ void collision_or_2d(bool* vp_collision, const bool* ray_tri_collision, size_t n_vp, size_t n_tri) {
    // for each viewpoint-triangle correspondance, check if rays to each vertex collide with any other triangle. if so, write in true
    // get viewpoint index
    size_t vp_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_vp

    if (vp_idx > n_vp - 1) { return; } // n_tri = n_vp

    vp_collision[vp_idx] = false;
    for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
        size_t res_idx = tri_idx * n_vp + vp_idx;
        if (ray_tri_collision[res_idx]) {
            vp_collision[vp_idx] = true;
        }
    }
    return;
}

__global__ void inc_angle(
    float *angles, // flattened 2d
    const vec3 *poses, // n_vp (vec3)
    const vec3 *centroids, // n_tri (vec3)
    const vec3 *normals, // n_tri (vec3)
    size_t n_vp,
    size_t n_tri
    ) {

    // get indices
    size_t vp_idx = blockIdx.x * blockDim.x + threadIdx.x; // n_vp
    size_t tri_idx = blockIdx.y * blockDim.y + threadIdx.y; // n_tri
    size_t res_idx = tri_idx * n_vp + vp_idx; // n_tri * n_vp

    if (vp_idx > n_vp - 1 || tri_idx > n_tri - 1) { return; }

    // calculate angle
    // vec3 vec = centroids[tri_idx] - poses[vp_idx];
    vec3 vec = poses[vp_idx] - centroids[tri_idx];
    vec3 norm = normals[tri_idx];
    float angle = acos(vec.dot(norm)/(vec.norm()*norm.norm()));
    angles[res_idx] = angle;
    return;
}

extern "C" void cuda_kernel_coverage(
    const Viewpoint& vp, 
    const std::vector<Triangle*>& faces,
    std::vector<bool>& collisions,
    vec3** int_points
    ) {

    // put viewpoints into arrays
    size_t n_tri = faces.size();
    size_t n_ray = n_tri * 3;

    vec3 *ends = new vec3[n_ray];
    Triangle *tri = new Triangle[n_tri];
    bool *result_arr = new bool[n_tri];
    bool *intersection_arr = new bool[n_ray * n_tri];
    vec3 *result_int_points = new vec3[n_ray * n_tri];

    // thread, block size
    size_t thread_x = 16;
    size_t thread_y = 16;
    size_t thread_z = 3;

    // put faces into array
    for (size_t i = 0; i < n_tri; i++) {
        tri[i] = *faces[i];
    }

    // put viewpoints into array
    for (size_t tri_pt_idx = 0; tri_pt_idx < 3; tri_pt_idx++) {
        for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
            size_t ray_idx = tri_idx * 3 + tri_pt_idx; // n_ray
            if (tri_pt_idx == 0) {
                ends[ray_idx] = tri[tri_idx].a;
            } else if (tri_pt_idx == 1) {
                ends[ray_idx] = tri[tri_idx].b;
            } else if (tri_pt_idx == 2) {
                ends[ray_idx] = tri[tri_idx].c;
            }
        }
    }

    // allocate gpu memory
    vec3 *d_ends;
    Triangle *d_tri;

    bool *d_intersections; // collisions per ray per triangle
    bool *d_result; // collisions per triangle
    vec3 *d_int_points; // intersection points

    hipMalloc(&d_ends, n_ray * sizeof(vec3));
    hipMalloc(&d_tri, n_tri * sizeof(Triangle));
    hipMalloc(&d_result, n_tri * sizeof(bool));
    hipMalloc(&d_intersections, n_ray * n_tri * sizeof(bool));
    hipMalloc(&d_int_points, n_ray * n_tri * sizeof(vec3));

    // copy data to gpu
    hipMemcpy(d_ends, ends, n_ray * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_tri, tri, n_tri * sizeof(Triangle), hipMemcpyHostToDevice);

    // set thread and block size
    dim3 threadsPerBlock(thread_x, thread_y, thread_z);

    // 2D blocks because 3d blocks can account for the 3rd dim by themselves
    dim3 numBlocks(int((n_tri + threadsPerBlock.x - 1) / threadsPerBlock.x), (n_tri + threadsPerBlock.y - 1) / threadsPerBlock.y, 1);
    ray_int_tri<<<numBlocks, threadsPerBlock>>>(
        d_intersections, 
        d_int_points, 
        vp.pose, 
        vp.viewdir,
        d_ends, 
        d_tri, 
        n_tri
    );

    hipMemcpy(intersection_arr, d_intersections, n_ray * n_tri * sizeof(bool), hipMemcpyDeviceToHost);

    // same as above but without the 3rd dimension
    threadsPerBlock.x = 1024;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;
    numBlocks.x = (n_tri + threadsPerBlock.x - 1) / threadsPerBlock.x;
    collision_or<<<numBlocks, threadsPerBlock>>>(d_result, d_intersections, n_tri, n_tri);

    hipMemcpy(result_arr, d_result, n_tri * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(result_int_points, d_int_points, n_ray * n_tri * sizeof(vec3), hipMemcpyDeviceToHost);

    hipFree(d_ends);
    hipFree(d_tri);
    hipFree(d_result);
    hipFree(d_intersections);
    hipFree(d_int_points);

    for (size_t vp_idx=0; vp_idx < n_tri; vp_idx++) {
        collisions.push_back(result_arr[vp_idx]);
    }

    if (int_points != nullptr) {
        for (size_t vp_idx = 0; vp_idx < n_tri; vp_idx++) {
            for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
                for (size_t tri_pt_idx = 0; tri_pt_idx < 3; tri_pt_idx++) {
                    size_t res_idx = tri_pt_idx * n_tri * n_tri + tri_idx * n_tri + vp_idx;
                    size_t ray_idx = tri_idx * 3 + tri_pt_idx;
                    if (intersection_arr[res_idx]) {
                        int_points[vp_idx][ray_idx] = result_int_points[res_idx];
                    } else {
                        int_points[vp_idx][ray_idx] = vec3(
                            std::numeric_limits<float>::infinity(), 
                            std::numeric_limits<float>::infinity(), 
                            std::numeric_limits<float>::infinity()
                        );
                    }
                    // int_points[vp_idx][ray_idx] = result_int_points[res_idx];
                }
            }
        }
    }

    delete[] ends;
    delete[] tri;
    delete[] result_arr;
    delete[] intersection_arr;
    delete[] result_int_points;
}

extern "C" void cuda_kernel_ray_int_plane(
    const std::vector<vec3>& ray_starts,
    const std::vector<vec3>& ray_ends,
    const vec3& plane_point,
    const vec3& plane_normal,
    bool* collisions,
    vec3* int_points
) {
    size_t n_rays = ray_starts.size();

    vec3 *starts = new vec3[n_rays];
    vec3 *ends = new vec3[n_rays];
    bool *result_arr = new bool[n_rays];
    vec3 *result_int_points = new vec3[n_rays];

    // thread, block size
    size_t thread_x = 1024;

    // put viewpoints into array
    for (size_t i = 0; i < n_rays; i++) {
        starts[i] = ray_starts[i];
        ends[i] = ray_ends[i];
    }

    vec3 *d_starts;
    vec3 *d_ends;
    bool *d_result;
    vec3 *d_int_points;

    hipMalloc(&d_starts, n_rays * sizeof(vec3));
    hipMalloc(&d_ends, n_rays * sizeof(vec3));
    hipMalloc(&d_result, n_rays * sizeof(bool));
    hipMalloc(&d_int_points, n_rays * sizeof(vec3));

    hipMemcpy(d_starts, starts, n_rays * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_ends, ends, n_rays * sizeof(vec3), hipMemcpyHostToDevice);

    // set up thread and block size
    dim3 threadsPerBlock(thread_x, 1, 1);
    dim3 numBlocks((n_rays + threadsPerBlock.x - 1) / threadsPerBlock.x, 1, 1);

    // run kernel
    ray_int_plane<<<numBlocks, threadsPerBlock>>>(
        d_result,
        d_int_points,
        d_starts,
        d_ends,
        plane_point,
        plane_normal,
        n_rays
    );

    hipDeviceSynchronize();

    // copy results back to host
    hipMemcpy(result_arr, d_result, n_rays * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(result_int_points, d_int_points, n_rays * sizeof(vec3), hipMemcpyDeviceToHost);

    // free gpu memory
    hipFree(d_starts);
    hipFree(d_ends);
    hipFree(d_result);
    hipFree(d_int_points);

    // copy memory into output
    for (size_t i = 0; i < n_rays; i++) {
        collisions[i] = result_arr[i];
        int_points[i] = result_int_points[i];
    }

    delete[] starts;
    delete[] ends;
    delete[] result_arr;
    delete[] result_int_points;
}

extern "C" void cuda_kernel_many_ray(
    const std::vector<vec3>& start_ray,
    const std::vector<vec3>& end_ray,
    const std::vector<Triangle*>& faces,
    bool* collisions,
    vec3** int_points
    ) {

    // put viewpoints into arrays
    size_t n_tri = faces.size();
    size_t n_ray = start_ray.size();

    vec3 *starts = new vec3[n_ray];
    vec3 *ends = new vec3[n_ray];
    Triangle *tri = new Triangle[n_tri];
    bool *result_arr = new bool[n_ray];
    bool *intersection_arr = new bool[n_ray * n_tri];
    vec3 *result_int_points = new vec3[n_ray * n_tri];

    // thread, block size
    size_t thread_x = 32;
    size_t thread_y = 32;
    size_t thread_z = 1;

    // put faces into array
    for (size_t i = 0; i < n_tri; i++) {
        tri[i] = *faces[i];
    }

    // put viewpoints into array
    for (size_t ray_idx = 0; ray_idx < n_ray; ray_idx++) {
        starts[ray_idx] = start_ray[ray_idx];
        ends[ray_idx] = end_ray[ray_idx];
    }

    // allocate gpu memory
    vec3 *d_starts;
    vec3 *d_ends;
    Triangle *d_tri;

    bool *d_intersections; // collisions per ray per triangle
    bool *d_result; // collisions per triangle
    vec3 *d_int_points; // intersection points

    hipMalloc(&d_starts, n_ray * sizeof(vec3));
    hipMalloc(&d_ends, n_ray * sizeof(vec3));
    hipMalloc(&d_tri, n_tri * sizeof(Triangle));
    hipMalloc(&d_intersections, n_ray * n_tri * sizeof(bool));
    hipMalloc(&d_result, n_ray * sizeof(bool));
    hipMalloc(&d_int_points, n_ray * n_tri * sizeof(vec3));

    // copy data to gpu
    hipMemcpy(d_starts, starts, n_ray * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_ends, ends, n_ray * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_tri, tri, n_tri * sizeof(Triangle), hipMemcpyHostToDevice);

    // set thread and block size
    dim3 threadsPerBlock(thread_x, thread_y, thread_z);

    // 2D blocks because 3d blocks can account for the 3rd dim by themselves
    dim3 numBlocks(int((n_ray + threadsPerBlock.x - 1) / threadsPerBlock.x), (n_tri + threadsPerBlock.y - 1) / threadsPerBlock.y, 1);
    ray_int_tri_many_2d<<<numBlocks, threadsPerBlock>>>(
        d_intersections, 
        d_int_points,
        d_starts, 
        d_ends, 
        n_ray,
        d_tri, 
        n_tri
    );

    hipDeviceSynchronize();

    hipMemcpy(intersection_arr, d_intersections, n_ray * n_tri * sizeof(bool), hipMemcpyDeviceToHost);

    // same as above but without the 3rd dimension
    threadsPerBlock.x = 1024;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    // reusing numBlocks
    numBlocks.x = (n_tri + threadsPerBlock.x - 1) / threadsPerBlock.x;
    numBlocks.y = 1; // numBlocks.z is already 1
    collision_or_2d<<<numBlocks, threadsPerBlock>>>(d_result, d_intersections, n_ray, n_tri);

    hipDeviceSynchronize();

    hipMemcpy(result_arr, d_result, n_ray * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(result_int_points, d_int_points, n_ray * n_tri * sizeof(vec3), hipMemcpyDeviceToHost);

    hipFree(d_starts);
    hipFree(d_ends);
    hipFree(d_tri);
    hipFree(d_result);
    hipFree(d_intersections);
    hipFree(d_int_points);

    for (size_t vp_idx=0; vp_idx < n_ray; vp_idx++) {
        collisions[vp_idx] = result_arr[vp_idx];
    }

    if (int_points != nullptr) {
        for (size_t ray_idx = 0; ray_idx < n_ray; ray_idx++){
            for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
                size_t res_idx = tri_idx * n_ray + ray_idx;
                int_points[ray_idx][tri_idx] = result_int_points[res_idx];
            }
        }
    }

    delete[] starts;
    delete[] ends;
    delete[] tri;
    delete[] result_arr;
    delete[] intersection_arr;
    delete[] result_int_points;
}


extern "C" void cuda_kernel_many(
    const std::vector<Viewpoint>& viewpoints,
    const std::vector<size_t>& triangle_indices,
    const std::vector<Triangle*>& faces,
    bool* collisions,
    vec3** int_points
    ) {

    // put viewpoints into arrays
    size_t n_tri = faces.size();
    size_t n_vp = viewpoints.size();
    size_t n_ray = n_vp * 3;

    vec3 *starts = new vec3[n_ray];
    vec3 *viewdirs = new vec3[n_ray];
    vec3 *ends = new vec3[n_ray];
    Triangle *tri = new Triangle[n_tri];
    bool *result_arr = new bool[n_vp];
    bool *intersection_arr = new bool[n_ray * n_tri];
    vec3 *result_int_points = new vec3[n_ray * n_tri];

    // thread, block size
    size_t thread_x = 16;
    size_t thread_y = 16;
    size_t thread_z = 3;

    // put faces into array
    for (size_t i = 0; i < n_tri; i++) {
        tri[i] = *faces[i];
    }

    // put viewpoints into array
    for (size_t tri_pt_idx = 0; tri_pt_idx < 3; tri_pt_idx++) {
        for (size_t vp_idx = 0; vp_idx < n_vp; vp_idx++) {
            size_t ray_idx = vp_idx * 3 + tri_pt_idx; // n_ray
            size_t tri_idx = triangle_indices[vp_idx];
            starts[ray_idx] = viewpoints[vp_idx].pose;
            viewdirs[ray_idx] = viewpoints[vp_idx].viewdir;
            if (tri_pt_idx == 0) {
                ends[ray_idx] = tri[tri_idx].a;
            } else if (tri_pt_idx == 1) {
                ends[ray_idx] = tri[tri_idx].b;
            } else if (tri_pt_idx == 2) {
                ends[ray_idx] = tri[tri_idx].c;
            }
        }
    }

    // allocate gpu memory
    vec3 *d_starts;
    vec3 *d_viewdirs;
    vec3 *d_ends;
    Triangle *d_tri;

    bool *d_intersections; // collisions per ray per triangle
    bool *d_result; // collisions per triangle
    vec3 *d_int_points; // intersection points

    hipMalloc(&d_starts, n_ray * sizeof(vec3));
    hipMalloc(&d_viewdirs, n_ray * sizeof(vec3));
    hipMalloc(&d_ends, n_ray * sizeof(vec3));
    hipMalloc(&d_tri, n_tri * sizeof(Triangle));
    hipMalloc(&d_intersections, n_ray * n_tri * sizeof(bool));
    hipMalloc(&d_result, n_vp * sizeof(bool));
    hipMalloc(&d_int_points, n_ray * n_tri * sizeof(vec3));

    // copy data to gpu
    hipMemcpy(d_starts, starts, n_ray * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_viewdirs, viewdirs, n_ray * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_ends, ends, n_ray * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_tri, tri, n_tri * sizeof(Triangle), hipMemcpyHostToDevice);

    // set thread and block size
    dim3 threadsPerBlock(thread_x, thread_y, thread_z);

    // 2D blocks because 3d blocks can account for the 3rd dim by themselves
    dim3 numBlocks(int((n_vp + threadsPerBlock.x - 1) / threadsPerBlock.x), (n_tri + threadsPerBlock.y - 1) / threadsPerBlock.y, 1);
    ray_int_tri_many<<<numBlocks, threadsPerBlock>>>(
        d_intersections, 
        d_int_points, 
        d_starts, 
        d_viewdirs,
        d_ends, 
        n_vp,
        d_tri, 
        n_tri
    );

    hipDeviceSynchronize();

    hipMemcpy(intersection_arr, d_intersections, n_ray * n_tri * sizeof(bool), hipMemcpyDeviceToHost);

    // same as above but without the 3rd dimension
    threadsPerBlock.x = 1024;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    // reusing numBlocks
    numBlocks.x = (n_tri + threadsPerBlock.x - 1) / threadsPerBlock.x;
    numBlocks.y = 1; // numBlocks.z is already 1
    collision_or<<<numBlocks, threadsPerBlock>>>(d_result, d_intersections, n_vp, n_tri);

    hipDeviceSynchronize();

    hipMemcpy(result_arr, d_result, n_vp * sizeof(bool), hipMemcpyDeviceToHost);
    hipMemcpy(result_int_points, d_int_points, n_ray * n_tri * sizeof(vec3), hipMemcpyDeviceToHost);

    hipFree(d_starts);
    hipFree(d_viewdirs);
    hipFree(d_ends);
    hipFree(d_tri);
    hipFree(d_result);
    hipFree(d_intersections);
    hipFree(d_int_points);

    for (size_t vp_idx=0; vp_idx < n_vp; vp_idx++) {
        collisions[vp_idx] = result_arr[vp_idx];
    }

    if (int_points != nullptr) {
        for (size_t vp_idx = 0; vp_idx < n_vp; vp_idx++) {
            for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
                for (size_t tri_pt_idx = 0; tri_pt_idx < 3; tri_pt_idx++) {
                    size_t res_idx = tri_pt_idx * n_tri * n_vp + tri_idx * n_vp + vp_idx;
                    size_t ray_idx = tri_idx * 3 + tri_pt_idx;
                    if (intersection_arr[res_idx]) {
                        int_points[vp_idx][ray_idx] = result_int_points[res_idx];
                    } else {
                        int_points[vp_idx][ray_idx] = vec3(
                            std::numeric_limits<float>::infinity(), 
                            std::numeric_limits<float>::infinity(), 
                            std::numeric_limits<float>::infinity()
                        );
                    }
                }
            }
        }
    }

    delete[] starts;
    delete[] viewdirs;
    delete[] ends;
    delete[] tri;
    delete[] result_arr;
    delete[] intersection_arr;
    delete[] result_int_points;
}

extern "C" void cuda_kernel_collision_points(
    const std::vector<Viewpoint>& viewpoints,
    const std::vector<Triangle*>& faces,
    const vec3 free_space_point,
    std::vector<bool>& in_collision // number of collisions
    ) {

    // put viewpoints into arrays
    size_t n_tri = faces.size();
    size_t n_vp = viewpoints.size();

    vec3 *starts = new vec3[n_vp];
    // vec3 *viewdirs = new vec3[n_vp];
    vec3 *ends = new vec3[n_vp];
    Triangle *tri = new Triangle[n_tri];
    bool *result_arr = new bool[n_vp];
    bool *intersection_arr = new bool[n_vp * n_tri];
    vec3 *result_int_points = new vec3[n_vp * n_tri];

    // thread, block size
    size_t thread_x = 32;
    size_t thread_y = 32;
    size_t thread_z = 1;

    // put faces into array
    for (size_t i = 0; i < n_tri; i++) {
        tri[i] = *faces[i];
    }

    // put viewpoints into array
    for (size_t vp_idx = 0; vp_idx < n_vp; vp_idx++) {
        starts[vp_idx] = viewpoints[vp_idx].pose;
        // viewdirs[vp_idx] = viewpoints[vp_idx].viewdir;
        ends[vp_idx] = free_space_point;
    }

    // allocate gpu memory
    vec3 *d_starts;
    // vec3 *d_viewdirs;
    vec3 *d_ends;
    Triangle *d_tri;

    bool *d_intersections; // collisions per ray per triangle
    bool *d_result; // collisions per triangle
    vec3 *d_int_points; // intersection points

    hipMalloc(&d_starts, n_vp * sizeof(vec3));
    // hipMalloc(&d_viewdirs, n_vp * sizeof(vec3));
    hipMalloc(&d_ends, n_vp * sizeof(vec3));
    hipMalloc(&d_tri, n_tri * sizeof(Triangle));
    hipMalloc(&d_intersections, n_vp * n_tri * sizeof(bool));
    hipMalloc(&d_result, n_vp * sizeof(bool));
    hipMalloc(&d_int_points, n_vp * n_tri * sizeof(vec3));

    // copy data to gpu
    hipMemcpy(d_starts, starts, n_vp * sizeof(vec3), hipMemcpyHostToDevice);
    // hipMemcpy(d_viewdirs, viewdirs, n_vp * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_ends, ends, n_vp * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_tri, tri, n_tri * sizeof(Triangle), hipMemcpyHostToDevice);

    // set thread and block size
    dim3 threadsPerBlock(thread_x, thread_y, thread_z);

    // 2D blocks because 3d blocks can account for the 3rd dim by themselves
    dim3 numBlocks(int((n_vp + threadsPerBlock.x - 1) / threadsPerBlock.x), (n_tri + threadsPerBlock.y - 1) / threadsPerBlock.y, 1);
    ray_int_tri_many_2d<<<numBlocks, threadsPerBlock>>>(
        d_intersections, 
        d_int_points, 
        d_starts, 
        // d_viewdirs,
        d_ends,
        n_vp,
        d_tri, 
        n_tri
    );

    hipDeviceSynchronize();

    hipMemcpy(intersection_arr, d_intersections, n_vp * n_tri * sizeof(bool), hipMemcpyDeviceToHost);
    // hipMemcpy(result_int_points, d_int_points, n_vp * n_tri * sizeof(vec3), hipMemcpyDeviceToHost);

    // same as above but without the 3rd dimension
    threadsPerBlock.x = 1024;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    // reusing numBlocks
    numBlocks.x = (n_tri + threadsPerBlock.x - 1) / threadsPerBlock.x;
    numBlocks.y = 1; // numBlocks.z is already 1
    collision_odd<<<numBlocks, threadsPerBlock>>>(
        d_result, 
        d_intersections, 
        n_vp, 
        n_tri
    );

    hipDeviceSynchronize();

    hipMemcpy(result_arr, d_result, n_vp * sizeof(bool), hipMemcpyDeviceToHost);

    hipFree(d_starts);
    // hipFree(d_viewdirs);
    hipFree(d_ends);
    hipFree(d_tri);
    hipFree(d_result);
    hipFree(d_intersections);
    hipFree(d_int_points);

    for (size_t vp_idx=0; vp_idx < n_vp; vp_idx++) {
        in_collision.push_back(result_arr[vp_idx]);
    }

    // if (int_points != nullptr) {
    //     for (size_t vp_idx = 0; vp_idx < n_vp; vp_idx++) {
    //         for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
    //             for (size_t tri_pt_idx = 0; tri_pt_idx < 3; tri_pt_idx++) {
    //                 size_t res_idx = tri_pt_idx * n_tri * n_vp + tri_idx * n_vp + vp_idx;
    //                 size_t ray_idx = tri_idx * 3 + tri_pt_idx;
    //                 if (intersection_arr[res_idx]) {
    //                     int_points[vp_idx][ray_idx] = result_int_points[res_idx];
    //                 } else {
    //                     int_points[vp_idx][ray_idx] = vec3(
    //                         std::numeric_limits<float>::infinity(), 
    //                         std::numeric_limits<float>::infinity(), 
    //                         std::numeric_limits<float>::infinity()
    //                     );
    //                 }
    //             }
    //         }
    //     }
    // }

    delete[] starts;
    // delete[] viewdirs;
    delete[] ends;
    delete[] tri;
    delete[] result_arr;
    delete[] intersection_arr;
    delete[] result_int_points;
}

extern "C" void cuda_kernel_inc_angle(
    const std::vector<Viewpoint>& viewpoints,
    const std::vector<Triangle*>& faces,
    std::vector<std::vector<float>>& inc_angles // n_vp x n_tri
    ) {

    // put viewpoints into arrays
    size_t n_tri = faces.size();
    size_t n_vp = viewpoints.size();

    vec3 *poses = new vec3[n_vp];
    vec3 *centroids = new vec3[n_tri];
    vec3 *normals = new vec3[n_tri];
    float *angles = new float[n_vp * n_tri];

    // put faces into array
    for (size_t vp_idx = 0; vp_idx < n_vp; vp_idx++) {
        poses[vp_idx] = viewpoints[vp_idx].pose;
    }

    for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
        centroids[tri_idx] = faces[tri_idx]->getCentroid();
        normals[tri_idx] = faces[tri_idx]->n;
    }

    // initialize pointers for gpu memory
    vec3 *d_poses;
    vec3 *d_centroids;
    vec3 *d_normals;
    float *d_angles;

    // allocate memory on gpu
    hipMalloc(&d_poses, n_vp * sizeof(vec3));
    hipMalloc(&d_centroids, n_tri * sizeof(vec3));
    hipMalloc(&d_normals, n_tri * sizeof(vec3));
    hipMalloc(&d_angles, n_vp * n_tri * sizeof(float));

    // copy data to gpu
    hipMemcpy(d_poses, poses, n_vp * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_centroids, centroids, n_tri * sizeof(vec3), hipMemcpyHostToDevice);
    hipMemcpy(d_normals, normals, n_tri * sizeof(vec3), hipMemcpyHostToDevice);

    // set thread and block size
    size_t thread_x = 32;
    size_t thread_y = 32;
    size_t thread_z = 1;
    dim3 threadsPerBlock(thread_x, thread_y, thread_z);
    dim3 numBlocks(
        int((n_vp + threadsPerBlock.x - 1) / threadsPerBlock.x), 
        int((n_tri + threadsPerBlock.y - 1) / threadsPerBlock.y),
        1
    );

    // calculate incidence angles
    inc_angle<<<numBlocks, threadsPerBlock>>>(
        d_angles, 
        d_poses, 
        d_centroids, 
        d_normals, 
        n_vp, 
        n_tri
    );

    // copy data from gpu
    hipMemcpy(angles, d_angles, n_vp * n_tri * sizeof(float), hipMemcpyDeviceToHost);

    // free memory from gpu
    hipFree(d_poses);
    hipFree(d_centroids);
    hipFree(d_normals);
    hipFree(d_angles);

    // put data into inc_angles
    for (size_t vp_idx = 0; vp_idx < n_vp; vp_idx++) {
        std::vector<float> vp_angles;
        for (size_t tri_idx = 0; tri_idx < n_tri; tri_idx++) {
            vp_angles.push_back(angles[tri_idx * n_vp + vp_idx]);
        }
        inc_angles.push_back(vp_angles);
    }

    delete[] poses;
    delete[] centroids;
    delete[] normals;
    delete[] angles;
}